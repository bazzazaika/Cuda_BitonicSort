#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <intrin.h>

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <algorithm>

//#define SIZE 536870912
int SIZE;
//2^12 4096
//2^20 1048576
//2^25 33554432
// 2^26 67108864
// 2^27 134217728
// 2^28 268435456

void generate(int* arr, int length)
{

	srand(time(NULL));
	unsigned int i;
	for (i = 0; i < length; ++i)
	{
		arr[i] = rand();
	}
}

__device__ void swap(int* one, int* two)
{
	int temp = *one;
	*one = *two;
	*two = temp;
}

__global__ void Ker_bitonicSort(int* mas, int k, int j, int SIZE)
{
	unsigned int i;
	i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < SIZE)
	{
		int ind = i | j;//узнаем индекс элемента, с которым хотим сравнивать
		if ((i & k) == 0)
		{
			if (mas[i] > mas[ind])
			{
				swap(&mas[i], &mas[ind]);
			}
		}
		else
		{
			if (mas[i] < mas[ind])
			{
				swap(&mas[i], &mas[ind]);
			}
		}
	}
}


void printArr(int* dev_values, int j, int k)
{
	printf("j = %d, k = %d\n", j, k);
	for (int i = 0; i < SIZE; i++)
	{
		printf("%d ", dev_values[i]);
	}
	printf("\n\n");
}

void bitonicSort(int* mas)
{
	int* cuda_mas;
	size_t size = SIZE * sizeof(int);
	hipMalloc((void**)&cuda_mas, size);
	hipMemcpy(cuda_mas, mas, size, hipMemcpyHostToDevice);
	int threads = 1024;
	int blocks = (SIZE / threads == 0) ? 1 : SIZE / threads;
	int j, k;

	for (k = 2; k <= SIZE; k <<= 1)
	{
		for (j = k >> 1; j > 0; j = j >> 1)
		{
			Ker_bitonicSort << <blocks, threads >> > (cuda_mas, k, j, SIZE);
			//hipMemcpy(mas, cuda_mas, size, hipMemcpyDeviceToHost);
			//printArr(mas, j, k);
		}
	}
	hipMemcpy(mas, cuda_mas, size, hipMemcpyDeviceToHost);
	hipFree(cuda_mas);
	//printf("Done on GPU!\n");
}

int check(int* values)
{
	int i;
	for (i = 0; i < SIZE - 1;i++)
	{
		if (values[i] > values[i + 1])
			return 0;
	}
	//printf("Good!\n");
	return 1;
}
int main(void)
{
	unsigned long long start, stop;

	//int mas[8] = { 4,3,9,5,6,2,1,7 };
	//printArr(mas, 0, 0);
	SIZE = 268435456;//268435456
	printf("SIZE = %d\n", SIZE);
	int* mas = (int*)malloc(SIZE * sizeof(int));
	generate(mas, SIZE);
	//printArr(mas, 0, 0);
	//start = __rdtsc();
	bitonicSort(mas);
	//stop = __rdtsc();
	if (check(mas))
		printf("Good!\n");
		//printf("TIME = %llu\n", (stop - start));


}